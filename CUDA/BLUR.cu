#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#define BLUR_SIZE 11
#define CHANNELS 3

__global__
void colorToBlurConversion(unsigned char* Pout, unsigned
	char* Pin, int width, int height) {

	int Col = blockIdx.x * blockDim.x + threadIdx.x;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	if (Col < width && Row < height) {
		int pixValsr = 0;
		int pixValsg = 0;
		int pixValsb = 0;
		int pixels=0;
		int Offset = (Row * width + Col) * CHANNELS;

		for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
			for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
				int curRow = Row + blurRow;
				int curCol = Col + blurCol;
				if (curRow > -1 && curRow < height && curCol > -1 && curCol < width) {
					int curOffset = (curRow * width + curCol) * CHANNELS;
					pixValsr += Pin[curOffset];
					pixValsg += Pin[curOffset + 1];
					pixValsb += Pin[curOffset + 2];
					pixels++;
				}
			}
		}
		Pout[Offset] = (unsigned char)(pixValsr / pixels);
		Pout[Offset + 1] = (unsigned char)(pixValsg / pixels);
		Pout[Offset + 2] = (unsigned char)(pixValsb / pixels);
	}
}
int main(int arc, char** argv) {
	int width, height, rgb;
	unsigned char* Pin = stbi_load(argv[1], &width, &height, &rgb, 3);
	unsigned char* ptrImageData = NULL;
	unsigned char* ptrImageDataOut = NULL;
 	/*Reserva y asignacion de memoria de Host a Dispositivo*/
	hipMalloc(&ptrImageDataOut, width * height * CHANNELS);
	hipMalloc(&ptrImageData, width * height * CHANNELS);
	hipMemcpy(ptrImageData, Pin, width * height * CHANNELS, hipMemcpyHostToDevice);
    /*Invocacion de la funcion Kernel*/
	colorToBlurConversion << <dim3((width / 16), (height / 16)), dim3(16, 16) >> > (ptrImageDataOut, ptrImageData, width, height);
	/*Copia de memoria de dispositivo a Host*/
    hipMemcpy(Pin, ptrImageDataOut, width * height * CHANNELS, hipMemcpyDeviceToHost);
	std::string NewImageFile = argv[1];
	NewImageFile = NewImageFile.substr(0, NewImageFile.find_last_of('.')) + "out.png";
	stbi_write_png(NewImageFile.c_str(), width, height, 3, Pin, 3 * width);
	stbi_image_free(Pin);
    /*Liberacion de memoria*/
	hipFree(ptrImageData);
	hipFree(ptrImageDataOut);
}
